#include "hip/hip_runtime.h"
#include "model/yolov5.hpp"
#include <vector>
#include <memory>
#include "slice/slice.hpp"
#include "model/affine.hpp"
#include "common/check.hpp"

#ifdef TRT10
#include "common/tensorrt.hpp"
namespace TensorRT = TensorRT10;
#else
#include "common/tensorrt8.hpp"
namespace TensorRT = TensorRT8;
#endif

#define GPU_BLOCK_THREADS 512

namespace yolov5
{

static const int NUM_BOX_ELEMENT = 8;  // left, top, right, bottom, confidence, class, keepflag, row_index(output)
static const int MAX_IMAGE_BOXES = 1024;

static dim3 grid_dims(int numJobs){
  int numBlockThreads = numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
  return dim3(((numJobs + numBlockThreads - 1) / (float)numBlockThreads));
}

static dim3 block_dims(int numJobs){
  return numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
}

static __host__ __device__ void affine_project(float *matrix, float x, float y, float *ox, float *oy) 
{
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel_v5(float *predict, int num_bboxes, int num_classes,
                                              int output_cdim, float confidence_threshold,
                                              float *invert_affine_matrix, float *parray, int *box_count,
                                              int max_image_boxes, int start_x, int start_y) 
{
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes) return;

    float *pitem = predict + output_cdim * position;
    float objectness = pitem[4];
    if (objectness < confidence_threshold) return;

    float *class_confidence = pitem + 5;
    
    float confidence = *class_confidence++;
    int label = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) 
    {
        if (*class_confidence > confidence) 
        {
            confidence = *class_confidence;
            label = i;
        }
    }
    confidence *= objectness;
    if (confidence < confidence_threshold) return;

    int index = atomicAdd(box_count, 1);
    if (index >= max_image_boxes) return;

    float cx = *pitem++;
    float cy = *pitem++;
    float width = *pitem++;
    float height = *pitem++;
    float left = cx - width * 0.5f;
    float top = cy - height * 0.5f;
    float right = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;
    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    float *pout_item = parray + index * NUM_BOX_ELEMENT;
    *pout_item++ = left + start_x;
    *pout_item++ = top + start_y;
    *pout_item++ = right + start_x;
    *pout_item++ = bottom + start_y;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1;  // 1 = keep, 0 = ignore
    *pout_item++ = position;
}


static __device__ float box_iou(float aleft, float atop, float aright, float abottom, float bleft,
                                float btop, float bright, float bbottom)
{
    float cleft = max(aleft, bleft);
    float ctop = max(atop, btop);
    float cright = min(aright, bright);
    float cbottom = min(abottom, bbottom);

    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if (c_area == 0.0f) return 0.0f;

    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}


static __global__ void fast_nms_kernel(float *bboxes, int* box_count, int max_image_boxes, float threshold) 
{
    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    // int count = min((int)*box_count, MAX_IMAGE_BOXES);
    int count = max_image_boxes;
    if (position >= count) return;

    // left, top, right, bottom, confidence, class, keepflag
    float *pcurrent = bboxes + position * NUM_BOX_ELEMENT;
    for (int i = 0; i < count; ++i) 
    {
        float *pitem = bboxes + i * NUM_BOX_ELEMENT;
        if (i == position || pcurrent[5] != pitem[5]) continue;

        if (pitem[4] >= pcurrent[4]) 
        {
            if (pitem[4] == pcurrent[4] && i < position) continue;

            float iou = box_iou(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3], pitem[0], pitem[1],
                                pitem[2], pitem[3]);

            if (iou > threshold) 
            {
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
}

static void decode_kernel_invoker(float *predict, int num_bboxes, int num_classes, int output_cdim,
                                  float confidence_threshold, float nms_threshold,
                                  float *invert_affine_matrix, float *parray, int* box_count, int max_image_boxes,
                                  int start_x, int start_y, hipStream_t stream) 
{
    auto grid = grid_dims(num_bboxes);
    auto block = block_dims(num_bboxes);

    checkKernel(decode_kernel_v5<<<grid, block, 0, stream>>>(
            predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
            parray, box_count, max_image_boxes, start_x, start_y));

    // grid = grid_dims(MAX_IMAGE_BOXES);
    // block = block_dims(MAX_IMAGE_BOXES);
    // checkKernel(fast_nms_kernel<<<grid, block, 0, stream>>>(parray, box_count, MAX_IMAGE_BOXES, nms_threshold));
}

static void fast_nms_kernel_invoker(float *parray, int* box_count, int max_image_boxes, float nms_threshold, hipStream_t stream)
{
    auto grid = grid_dims(max_image_boxes);
    auto block = block_dims(max_image_boxes);
    checkKernel(fast_nms_kernel<<<grid, block, 0, stream>>>(parray, box_count, max_image_boxes, nms_threshold));
}

class Yolov5ModelImpl : public Infer 
{
public:
    // for sahi crop image
    std::shared_ptr<slice::SliceImage> slice_;
    std::shared_ptr<TensorRT::Engine> trt_;
    std::string engine_file_;

    tensor::Memory<int> box_count_;

    tensor::Memory<float> affine_matrix_;
    tensor::Memory<float>  input_buffer_, bbox_predict_, output_boxarray_;

    int network_input_width_, network_input_height_;
    affine::Norm normalize_;
    std::vector<int> bbox_head_dims_;
    bool isdynamic_model_ = false;

    float confidence_threshold_;
    float nms_threshold_;

    int num_classes_ = 0;

    virtual ~Yolov5ModelImpl() = default;

    void adjust_memory(int batch_size) 
    {
        // the inference batch_size
        size_t input_numel = network_input_width_ * network_input_height_ * 3;
        input_buffer_.gpu(batch_size * input_numel);
        bbox_predict_.gpu(batch_size * bbox_head_dims_[1] * bbox_head_dims_[2]);
        output_boxarray_.gpu(batch_size * (MAX_IMAGE_BOXES * NUM_BOX_ELEMENT));
        output_boxarray_.cpu(batch_size * (MAX_IMAGE_BOXES * NUM_BOX_ELEMENT));

        affine_matrix_.gpu(6);
        affine_matrix_.cpu(6);

        box_count_.gpu(1);
        box_count_.cpu(1);
    }

    void preprocess(int ibatch, affine::LetterBoxMatrix &affine, void *stream = nullptr)
    {
        affine.compute(std::make_tuple(slice_->slice_width_, slice_->slice_height_),
                    std::make_tuple(network_input_width_, network_input_height_));

        size_t input_numel = network_input_width_ * network_input_height_ * 3;
        float *input_device = input_buffer_.gpu() + ibatch * input_numel;
        size_t size_image = slice_->slice_width_ * slice_->slice_height_ * 3;

        float *affine_matrix_device = affine_matrix_.gpu();
        uint8_t *image_device = slice_->output_images_.gpu() + ibatch * size_image;

        float *affine_matrix_host = affine_matrix_.cpu();

        // speed up
        hipStream_t stream_ = (hipStream_t)stream;
        memcpy(affine_matrix_host, affine.d2i, sizeof(affine.d2i));
        checkRuntime(hipMemcpyAsync(affine_matrix_device, affine_matrix_host, sizeof(affine.d2i),
                                    hipMemcpyHostToDevice, stream_));

        affine::warp_affine_bilinear_and_normalize_plane(image_device, slice_->slice_width_ * 3, slice_->slice_width_,
                                                slice_->slice_height_, input_device, network_input_width_,
                                                network_input_height_, affine_matrix_device, 114,
                                                normalize_, stream_);
    }

    bool load(const std::string &engine_file, float confidence_threshold, float nms_threshold) 
    {
        trt_ = TensorRT::load(engine_file);
        if (trt_ == nullptr) return false;

        trt_->print();

        this->confidence_threshold_ = confidence_threshold;
        this->nms_threshold_ = nms_threshold;

        auto input_dim = trt_->static_dims(0);
        bbox_head_dims_ = trt_->static_dims(1);
        network_input_width_ = input_dim[3];
        network_input_height_ = input_dim[2];
        isdynamic_model_ = trt_->has_dynamic_dim();

        normalize_ = affine::Norm::alpha_beta(1 / 255.0f, 0.0f, affine::ChannelType::SwapRB);
        num_classes_ = bbox_head_dims_[2] - 5;
        return true;
    }


    virtual BoxArray forward(const tensor::Image &image, int slice_width, int slice_height, float overlap_width_ratio, float overlap_height_ratio, void *stream = nullptr) override 
    {
        slice_->slice(image, slice_width, slice_height, overlap_width_ratio, overlap_height_ratio, stream);
        return forwards(stream);
    }

    virtual BoxArray forward(const tensor::Image &image, void *stream = nullptr) override 
    {
        slice_->autoSlice(image, stream);
        return forwards(stream);
    }

    virtual BoxArray forwards(void *stream = nullptr) override 
    {
        int num_image = slice_->slice_num_h_ * slice_->slice_num_v_;
        if (num_image == 0) return {};
        
        auto input_dims = trt_->static_dims(0);
        int infer_batch_size = input_dims[0];
        if (infer_batch_size != num_image) 
        {
            if (isdynamic_model_) 
            {
                infer_batch_size = num_image;
                input_dims[0] = num_image;
                if (!trt_->set_run_dims(0, input_dims)) 
                {
                    printf("Fail to set run dims\n");
                    return {};
                }
            } 
            else 
            {
                if (infer_batch_size < num_image) 
                {
                    printf(
                        "When using static shape model, number of images[%d] must be "
                        "less than or equal to the maximum batch[%d].",
                        num_image, infer_batch_size);
                    return {};
                }
            }
        }
        adjust_memory(infer_batch_size);

        affine::LetterBoxMatrix affine_matrix;
        hipStream_t stream_ = (hipStream_t)stream;
        for (int i = 0; i < num_image; ++i)
            preprocess(i, affine_matrix, stream);

        float *bbox_output_device = bbox_predict_.gpu();
        #ifdef TRT10
        if (!trt_->forward(std::unordered_map<std::string, const void *>{
                { "images", input_buffer_.gpu() }, 
                { "output0", bbox_predict_.gpu() }
            }, stream_))
        {
            printf("Failed to tensorRT forward.");
            return {};
        }
        #else
        std::vector<void *> bindings{input_buffer_.gpu(), bbox_output_device};
        if (!trt_->forward(bindings, stream)) 
        {
            printf("Failed to tensorRT forward.");
            return {};
        }
        #endif

        int* box_count = box_count_.gpu();
        checkRuntime(hipMemsetAsync(box_count, 0, sizeof(int), stream_));
        for (int ib = 0; ib < num_image; ++ib) 
        {
            int start_x = slice_->slice_position_[ib*2];
            int start_y = slice_->slice_position_[ib*2+1];
            float *boxarray_device =
                output_boxarray_.gpu() + ib * (MAX_IMAGE_BOXES * NUM_BOX_ELEMENT);
            float *affine_matrix_device = affine_matrix_.gpu();
            float *image_based_bbox_output =
                bbox_output_device + ib * (bbox_head_dims_[1] * bbox_head_dims_[2]);
            decode_kernel_invoker(image_based_bbox_output, bbox_head_dims_[1], num_classes_,
                                    bbox_head_dims_[2], confidence_threshold_, nms_threshold_,
                                    affine_matrix_device, boxarray_device, box_count, MAX_IMAGE_BOXES, start_x, start_y, stream_);
        }
        // checkRuntime(hipStreamSynchronize(stream_));
        float *boxarray_device =  output_boxarray_.gpu();
        fast_nms_kernel_invoker(boxarray_device, box_count, MAX_IMAGE_BOXES * num_image, nms_threshold_, stream_);
        checkRuntime(hipMemcpyAsync(output_boxarray_.cpu(), output_boxarray_.gpu(),
                                    output_boxarray_.gpu_bytes(), hipMemcpyDeviceToHost, stream_));
        checkRuntime(hipMemcpyAsync(box_count_.cpu(), box_count_.gpu(),
                                    box_count_.gpu_bytes(), hipMemcpyDeviceToHost, stream_));
        checkRuntime(hipStreamSynchronize(stream_));

        BoxArray result;
        // int imemory = 0;
        for (int ib = 0; ib < num_image; ++ib) 
        {
            
            float *parray = output_boxarray_.cpu() + ib * (MAX_IMAGE_BOXES * NUM_BOX_ELEMENT);
            int count = min(MAX_IMAGE_BOXES, *(box_count_.cpu()));
            for (int i = 0; i < count; ++i) 
            {
                float *pbox = parray + i * NUM_BOX_ELEMENT;
                int label = pbox[5];
                int keepflag = pbox[6];
                if (keepflag == 1) {
                    Box result_object_box(pbox[0], pbox[1], pbox[2], pbox[3], pbox[4], label);
                    result.emplace_back(result_object_box);
                }
            }
        }
        return result;
    }

};


Infer *loadraw(const std::string &engine_file, float confidence_threshold,
               float nms_threshold) 
{
    Yolov5ModelImpl *impl = new Yolov5ModelImpl();
    if (!impl->load(engine_file, confidence_threshold, nms_threshold)) 
    {
        delete impl;
        impl = nullptr;
    }
    impl->slice_ = std::make_shared<slice::SliceImage>();
    return impl;
}

std::shared_ptr<Infer> load(const std::string &engine_file, int gpu_id, float confidence_threshold,
               float nms_threshold) 
{
    checkRuntime(hipSetDevice(gpu_id));
    return std::shared_ptr<Yolov5ModelImpl>((Yolov5ModelImpl *)loadraw(engine_file, confidence_threshold, nms_threshold));
}

}
